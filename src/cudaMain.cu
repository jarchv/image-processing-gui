#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <iostream>
#include <algorithm> 

#define CVUI_IMPLEMENTATION
#include "cvui.h"

#include "cudaMain.h"
#include "tools.h"
#include "base.h"

cv::Mat img;
cv::Mat img_res;
cv::Mat IMAGE;
cv::Mat frame = cv::Mat(720, 1280, CV_8UC3);
cv::Mat templ;
cv::Mat templ_res  = cv::Mat(60, 60, CV_8UC3,cv::Scalar(38, 36, 26));
cv::Mat chromCV;
cv::Mat grayCV;
cv::Mat swapGray;
cv::Mat trackResul;
cv::Mat TRACK;
cv::Mat RESULT;
cv::Mat frameCap;
cv::Mat inputbox;
cv::Mat imgFrame;

int cudaMain(int argc, char **argv)
{
    std::string X;
    
    if (argc == 1)
    {
        X = "files/24BITS.BMP";
        X = "../" + X;

        filename = X.c_str();        
        std::cout<<"\nFILE: "<<X<<"\n"<<std::endl;
        
        mdata = readBMPFile(filename, WIDTH, HEIGHT, DEPTH);
        img   = cv::Mat(cv::Size(WIDTH, HEIGHT), CV_8UC3, mdata);
    }
    else if (argc > 1)
    {        
        if (argc > 2)
        {
            if (argv[1][1] == 't')
            {
                X = argv[2];
                X = "../files/" + X; 
                
                filename = X.c_str();
                std::cout<<"\nFILE: "<<X<<"\n"<<std::endl;
                
                img     = cv::imread(filename,cv::IMREAD_COLOR );

                WIDTH   = (int)img.cols;
                HEIGHT  = (int)img.rows;
                mdata   = new unsigned char[WIDTH * HEIGHT * 3];
                //mdata   = img.data;
                mdata   = toArray(img);  
            }
        }
        else {
            X = argv[1];
            X = "../files/" + X; 
            
            filename = X.c_str();        
            std::cout<<"\nFILE: "<<X<<"\n"<<std::endl;
            mdata = readBMPFile(filename, WIDTH, HEIGHT, DEPTH);
            img   = cv::Mat(cv::Size(WIDTH, HEIGHT), CV_8UC3, mdata);
        }

    }

    else {
        printf("Unable to open file ...");
        return 0;
    }

    printf("Size = (%d, %d)\n", WIDTH,HEIGHT);
    
    cv::namedWindow(WINDOW_NAME);
    

    factor = getResizeFactor(WIDTH, HEIGHT, 480);


    cv::resize(img, img_res, cv::Size(), factor, factor, cv::INTER_LINEAR );
    cvui::init(WINDOW_NAME);

    toDisplay = new unsigned char[WIDTH * HEIGHT * 3];
    
    cv::VideoWriter video("out.avi", CV_FOURCC('M','J','P','G'),30, cv::Size(1280,720),true);

    while (true)
    {
        frame = cv::Scalar(38, 36, 26);     
        cvui::window(frame,  10,  10, 260, 330, "Settings");
        cvui::window(frame, 720,  10, 520, 680, "Result");
        cvui::window(frame, 400, 370, 300, 320, "Input");
        
        if (prev_iter != (int)iter)
            DONE = true;
        
        if (prev_brig != brig)
            DONE = true;

        if (prev_cont != cont)
            DONE = true; 

        switch (SET_CODE)
        {
            case 0: {
                img_res.copyTo(IMAGE);
                DONE = true;       
                break;
            }
            case 2: {
                if (DONE){
                    copy(mdata, toDisplay, WIDTH*HEIGHT*3);
                    for (int i = 0; i < (int)iter; i++)
                        toDisplay = meanFilter(toDisplay, WIDTH, HEIGHT);
                    imgFrame = cv::Mat(cv::Size(WIDTH, HEIGHT), CV_8UC3, toDisplay);
                    //cvtColor(imgFrame, imgFrame, cv::COLOR_BGR2RGB);
                    if (max(imgFrame.cols, imgFrame.rows) > MAX_DIM)
                        cv::resize(imgFrame, IMAGE, cv::Size(), factor, factor, cv::INTER_CUBIC);
                    else
                        imgFrame.copyTo(IMAGE);
                    DONE = false;
                    prev_iter = (int)iter;
                }
                break;
            }
            case 4: {
                if (DONE){
                    
                    copy(mdata, toDisplay, WIDTH*HEIGHT*3);
                    toDisplay = laplacianFilter(toDisplay, WIDTH, HEIGHT);
                    imgFrame = cv::Mat(cv::Size(WIDTH, HEIGHT), CV_8UC3, toDisplay);
                    
                    if (max(imgFrame.cols, imgFrame.rows) > MAX_DIM)
                        cv::resize(imgFrame, IMAGE, cv::Size(), factor, factor, cv::INTER_CUBIC);
                    else
                        imgFrame.copyTo(IMAGE);

                    cvtColor(IMAGE, IMAGE, cv::COLOR_RGB2GRAY);
                    cvtColor(IMAGE, IMAGE, cv::COLOR_GRAY2RGB);
                    DONE = false;   
                }
                break;
            }

            case 8: {
                if (DONE){
                    copy(mdata, toDisplay, WIDTH*HEIGHT*3);
                    toDisplay = sharpenFilter(toDisplay, WIDTH, HEIGHT);
                    imgFrame = cv::Mat(cv::Size(WIDTH, HEIGHT), CV_8UC3, toDisplay);
                    //cvtColor(imgFrame, imgFrame, cv::COLOR_BGR2RGB);

                    if (max(imgFrame.cols, imgFrame.rows) > MAX_DIM)
                        cv::resize(imgFrame, IMAGE, cv::Size(), factor, factor, cv::INTER_CUBIC);
                    else
                        imgFrame.copyTo(IMAGE);
                    DONE = false;   
                }
                break;
            }

            case 16: {
                if (DONE) {
                    grayimg = toGray(mdata, WIDTH, HEIGHT);
                    grayCV  = cv::Mat(cv::Size(WIDTH, HEIGHT), CV_8UC1, grayimg);
                    cvtColor(grayCV, grayCV, cv::COLOR_GRAY2RGB);
                    if (max(grayCV.cols, grayCV.rows) > MAX_DIM)
                        cv::resize(grayCV, IMAGE, cv::Size(), factor, factor, cv::INTER_CUBIC);
                    else
                        grayCV.copyTo(IMAGE);
                    DONE = false; 
                }
                break;
            }
            case 32: {
                if (DONE){
                    chromimg = toChromatic(mdata, WIDTH, HEIGHT);
                    chromCV  = cv::Mat(cv::Size(WIDTH, HEIGHT), CV_8UC3, chromimg);
                    if (max(chromCV.cols, chromCV.rows) > MAX_DIM)
                        cv::resize(chromCV, IMAGE, cv::Size(), factor, factor, cv::INTER_CUBIC);
                    else
                        chromCV.copyTo(IMAGE);
                    DONE = false;
                    prev_brig = brig;
                    prev_cont = cont;   
                }
                break;
            }
            case 64: {
                if (DONE){
                    copy(mdata, toDisplay, WIDTH*HEIGHT*3);
                    for (int i = 0; i < (int)iter; i++)
                        toDisplay = meanFilter(toDisplay, WIDTH, HEIGHT);
                    grayimg = toGray(toDisplay, WIDTH, HEIGHT);
                    img2fft = FFT(grayimg, HEIGHT, WIDTH);
                    imgBC   = BC(img2fft, (float)brig, cont, WIDTH * HEIGHT);

                    grayCV   = cv::Mat(cv::Size(WIDTH, HEIGHT), CV_8U, imgBC);
                    swapGray = fftSwap(grayCV, WIDTH, HEIGHT);
                    cvtColor(swapGray, swapGray, cv::COLOR_GRAY2RGB);

                    if (max(swapGray.cols, swapGray.rows) > MAX_DIM)
                        cv::resize(swapGray, IMAGE, cv::Size(), factor, factor, cv::INTER_CUBIC);
                    else
                        swapGray.copyTo(IMAGE);
                    prev_brig = brig;
                    prev_cont = cont; 
                    prev_iter = (int)iter;
                    DONE = false;
                }
                break;
            }
            case  128: {
                if (DONE)
                {
                    cv::Mat src;
                    img_res.copyTo(src);

                    std::vector<cv::Mat> bgr(3);
                    split(src,bgr);
                    
                    for (int i = 0; i < 3; i++)
                    {
                        bgr[i].convertTo(bgr[i],CV_32FC1);
                        HaarWavelet(bgr[i],2);

                        cv::normalize(bgr[i], bgr[i], 0, 250, cv::NORM_MINMAX, -1, cv::Mat());
                        bgr[i].convertTo(bgr[i], CV_8U);
                    }

                    cv::merge(bgr, IMAGE);
                    DONE = false;
                }
                break;
            }
            case 256: {
                cvui::window(frame, 620, 10,  80, 100, "Template");
                if (DONE){
                    templ    = cv::imread("../files/temp.png");
                    cv::VideoCapture cap("../files/videodemo.mp4");

                    if (!cap.isOpened())
                    {
                        std::cout << "Failed to open camera." << std::endl;
                    }          

                    else {
                        cap >> frameCap;
                        factor = getResizeFactor(frameCap.cols, frameCap.rows, 480);

                        int count = 0;
                        for(;;)
                        {
                            cvui::checkbox(frame, 30, 295, "Template Matching   ", &USE_TEMPLATE);
                            cap >> frameCap;
                            if(frameCap.empty())
                                break;
                            count++;
                            if(count > 128)
                                break;
                            if(USE_TEMPLATE == false)
                                break;

                            trackResul = TemplateMatching(frameCap, TRACK, templ);
                            cvtColor(trackResul, trackResul, cv::COLOR_GRAY2RGB);
                            cv::normalize( trackResul, trackResul, 0, 255, cv::NORM_MINMAX, -1, cv::Mat() );
                            trackResul.convertTo(trackResul, CV_8U);
                            if (max(TRACK.cols, TRACK.rows) > MAX_DIM){
                                cv::resize(TRACK, IMAGE, cv::Size(), factor, factor, cv::INTER_CUBIC);
                                cv::resize(trackResul, RESULT, cv::Size(), factor, factor, cv::INTER_CUBIC);
                            }
                            else{
                                TRACK.copyTo(IMAGE);
                                trackResul.copyTo(RESULT);
                            }

                            cv::resize(templ, templ_res, cv::Size(60,60), cv::INTER_CUBIC);
                            int xpos =  (int)((520 - IMAGE.cols)/2) + 720;
                            int ypos =  50+0;
                            Mat2Mat(IMAGE , frame, ypos, xpos);
                            Mat2Mat(RESULT, frame, ypos+300, xpos); 
                            Mat2Mat(templ_res, frame, 40, 630);

                            cv::imshow(WINDOW_NAME, frame);
                            video.write(frame);
                            k = cv::waitKey(1);
                            if (k == 27){
                                std::cout << "[ESC] : break" << std::endl;
                                break;
                            }                           
                        }
                        cap.release();                        
                    }

                    DONE = false;
                }
                factor = getResizeFactor(img.cols, img.rows, 480);
                break;
            }
            default:{
                DONE = true;
            }
        }

        

        //Mat2Mat(templ_res, frame, 40, 630);

        cvui::printf(frame,  20, 45, 0.4, 0xeeeeee, "Filters");

        if (cvui::checkbox(frame, 30, 65, "Mean Filter", &USE_MEAN_FILTER)){
            USE_LAPLACIAN_FILTER = false;
            USE_SHARPEN_FILTER   = false;
            USE_CHROMATIC        = false;
            USE_GRAY             = false;
            USE_FFT              = false;
            USE_TEMPLATE         = false;
            USE_WAVELET          = false;
        } 

        if (cvui::checkbox(frame, 30, 85, "Laplacian Filter", &USE_LAPLACIAN_FILTER)){
            USE_MEAN_FILTER      = false;
            USE_SHARPEN_FILTER   = false;
            USE_CHROMATIC        = false;
            USE_GRAY             = false;
            USE_FFT              = false;
            USE_TEMPLATE         = false;
            USE_WAVELET          = false;  
        }

        if (cvui::checkbox(frame, 30, 105, "Sharpen Filter", &USE_SHARPEN_FILTER)){
            USE_MEAN_FILTER      = false;
            USE_LAPLACIAN_FILTER = false;
            USE_CHROMATIC        = false;
            USE_GRAY             = false;
            USE_FFT              = false;
            USE_TEMPLATE         = false;
            USE_WAVELET          = false;
        }

        cvui::printf(frame,  20, 135, 0.4, 0xeeeeee, "Color Spaces");
        if (cvui::checkbox(frame, 30, 155, "Gray Scale", &USE_GRAY)){
            USE_MEAN_FILTER      = false;
            USE_LAPLACIAN_FILTER = false;
            USE_SHARPEN_FILTER   = false;
            USE_CHROMATIC        = false;
            USE_FFT              = false;
            USE_TEMPLATE         = false;
            USE_WAVELET          = false;
        }

        if (cvui::checkbox(frame, 30, 175, "Chromatic", &USE_CHROMATIC)){
            USE_MEAN_FILTER      = false;
            USE_LAPLACIAN_FILTER = false;
            USE_SHARPEN_FILTER   = false;
            USE_GRAY             = false;
            USE_FFT              = false;
            USE_WAVELET          = false;
            USE_TEMPLATE         = false;            
        }
        
        cvui::printf(frame,  20, 205, 0.4, 0xeeeeee, "Frecuency domain");
        if (cvui::checkbox(frame, 30, 225, "Fourier", &USE_FFT)){
            USE_MEAN_FILTER      = false;
            USE_LAPLACIAN_FILTER = false;
            USE_SHARPEN_FILTER   = false;
            USE_GRAY             = false;
            USE_CHROMATIC        = false;
            USE_WAVELET          = false;
            USE_TEMPLATE         = false;
        }
        if (cvui::checkbox(frame, 30, 245, "Wavelet", &USE_WAVELET)){
            USE_MEAN_FILTER      = false;
            USE_LAPLACIAN_FILTER = false;
            USE_SHARPEN_FILTER   = false;
            USE_GRAY             = false;
            USE_CHROMATIC        = false;
            USE_TEMPLATE         = false;
            USE_FFT              = false;
        }

        cvui::printf(frame,  20, 275, 0.4, 0xeeeeee, "Image Analysis");
        if (cvui::checkbox(frame, 30, 295, "Template Matching   ", &USE_TEMPLATE)){
            USE_MEAN_FILTER      = false;
            USE_LAPLACIAN_FILTER = false;
            USE_SHARPEN_FILTER   = false;
            USE_GRAY             = false;
            USE_CHROMATIC        = false;
            USE_FFT              = false;
            USE_WAVELET          = false;
        }

        SET_CODE =  USE_MEAN_FILTER      *  2  + 
                    USE_LAPLACIAN_FILTER *  4  +
                    USE_SHARPEN_FILTER   *  8  +
                    USE_GRAY             * 16  +
                    USE_CHROMATIC        * 32  +
                    USE_FFT              * 64  +
                    USE_WAVELET          * 128 +
                    USE_TEMPLATE         * 256;

        
        cvui::trackbar(frame, 828, 550, 300, &iter, 0.0, 20.0, 0.1, "",cvui::TRACKBAR_HIDE_LABELS); //"%1.Lf"
        cvui::trackbar(frame, 828, 585, 300, &brig,   0,  255,   1, "",cvui::TRACKBAR_HIDE_LABELS);
        cvui::trackbar(frame, 828, 615, 300, &cont, 0.1, 100.0, 0.1, "",cvui::TRACKBAR_HIDE_LABELS);

        cvui::printf(frame, 760 , 570, 0.4, 0xeeeeee, "Mean Filter");
        cvui::printf(frame, 760 , 605, 0.4, 0xeeeeee, "Brightness");
        cvui::printf(frame, 1120, 605, 0.4, 0xeeeeee, "(Fourier)");
        cvui::printf(frame, 760 , 635, 0.4, 0xeeeeee, "Contrast");
        cvui::printf(frame, 1120, 635, 0.4, 0xeeeeee, "(Fourier)");

        // 570, 605, 635
        cvui::printf(frame,  20, 350, 0.4, 0xeeeeee, "Filename : %s", filename);
        cvui::printf(frame,  20, 370, 0.4, 0xeeeeee, "Width  : %d", WIDTH);
        cvui::printf(frame,  20, 390, 0.4, 0xeeeeee, "Height : %d", HEIGHT);
        cvui::printf(frame,  20, 410, 0.4, 0xeeeeee, "Depth  : %d", DEPTH);

        if(cvui::button(frame, 10, 680, "&Quit")){
            break;
        }

        cvui::update();
        int xpos =  (int)((520 - IMAGE.cols)/2) + 720;
        int ypos =  50+0;
        Mat2Mat(IMAGE, frame, ypos, xpos);

        inputbox_factor = getResizeFactor(WIDTH, HEIGHT, 280);
        cv::resize(img, inputbox, cv::Size(), inputbox_factor, inputbox_factor, cv::INTER_CUBIC);
        Mat2Mat(inputbox, frame, 400, 410);
        
        
        cv::imshow(WINDOW_NAME, frame);
        video.write(frame);
        k = cv::waitKey(1);
        if (k == 27){
            std::cout << "[ESC] : break" << std::endl;
            break;
        }
    }  

    free(toDisplay);
    //free(mdata);
    free(grayimg);
    free(chromimg);
    free(img2fft);
    free(imgBC);

    video.release();
    //cvReleaseVideoWriter( &video );
    return 0;   
}